#include "hip/hip_runtime.h"
using namespace std;

__global__ void cuda_kernel(int *B, int *A, IndexSave *dInd)
{
	// complete cuda kernel function
	int i = 0;
	int N = 4*LOOP;
	int stripe = gridDim.x * blockDim.x;
	int head = (blockIdx.x*blockDim.x+threadIdx.x);
	for(i=head; i<N; i+=stripe)
	{
		B[i] = A[i] * A[i] * A[i] * A[i];
		dInd[i].blockInd_x = blockIdx.x;
		dInd[i].threadInd_x = threadIdx.x;
		dInd[i].head = head;
		dInd[i].stripe = stripe;
	}
};


float GPU_kernel(int *B,int *A,IndexSave* indsave){

	int *dA = 0,*dB;
	IndexSave* dInd;

	// Creat Timing Event
  hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	// Allocate Memory Space on Device
	int N = SIZE;																// Size of the memory
	hipMalloc((void**) &dB, sizeof(int)*N); 		// new a memory size in GPU for array B
	hipMalloc((void**) &dA, sizeof(int)*N); 		// new a memory size in GPU for array A



	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd, sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated
	hipMemcpy(dA, A, sizeof(int)*N, hipMemcpyHostToDevice); // copy array A to the  GPU
	// Copy Data (indsave array) to device, CPU to GPU
	hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);

	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel,method 1
	dim3 dimGrid(2);	// Block Size
	dim3 dimBlock(4); // Thread Size
	cuda_kernel<<<dimGrid,dimBlock>>>(dB, dA, dInd);
	hipDeviceSynchronize();
	// Stop Timer
	hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

	// Copy Output back
	hipMemcpy(B, dB, sizeof(int)*N, hipMemcpyDeviceToHost); // copy array A to the  GPU
	hipMemcpy(indsave, dInd, sizeof(IndexSave)*SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(A, dA, sizeof(int)*N, hipMemcpyDeviceToHost); // copy array A to the  GPU
	// Release Memory Space on Device
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	return elapsedTime;
}
